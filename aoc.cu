#include "hip/hip_runtime.h"
#define block_size = 4
#define raio = 2
#define n = 12

__global__ void foo(int *input, int *output ) {

  /*shared: compartilha com a memoria global*/
  __shared__ int temp[block_size+(raio*2)];
  int ind_global = blockIdx.x * blockDim.x + threadIdx.x;

  //copia o meio para temp
  input[ind_local] = threadIdx.x + raio;
  temp[ind_local] = input[ind_global];

  //copia os dois adjacentes para temp
  if(threadIdx.x < raio){
    if(ind_local - raio >= 0){
      temp[ind_local - raio] = input[ind_global - raio];
    }
    if(ind_local + block_size < n){
      temp[ind_local + block_size] = input[ind_global + block_size];
    }
  }

  __syncthreads();
  //atualizar as entradas
  int soma = 0;
  for(int i = 0; i < ind_local - raio; i++){
    if(ind_local - raio + i >= 0 && ind_local - raio + 1 <= n)
      soma += temp[ind_local - raio + i];
  }
    output[ind_global] = soma;
}

int main (void){
  int num_blocos = 3, num_threads = 4;
  int input[12] = {1,3,2,1,0,2,4,1,5,3,2,4};

  hipMalloc((void**)&dev_input, n*sizeof(int));
  hipMalloc((void**)&dev_output, n*sizeof(int));
  hipMemcpy(dev_input, input, n*sizeof(int));
  hipMemcpy(dev_output, output, n*sizeof(int), hipMemcpyHostToDevice);
  foo <<< num_blocos, num_threads >>> (dev_input, dev_output);
  hipMemcpy(output, dev_output, hipMemcpyDeviceToHost);

  for (int i = 0; i < n; i++){
    printf("%d\n", output[i]);
  }
}
//usr/local/cuda-8.0/bin/nvcc
